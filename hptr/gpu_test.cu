#include "hip/hip_runtime.h"
﻿#include "config.h"

#if GPU_TEST

#include "base.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__global__ void kernel(void)
{
	Hptr<A> hptr1(new A(1));
	hptr1->print();	// 测试->
	A r = *hptr1;	// 测试*
	r.print();

	// 测试=
	Hptr<A> hptr2(new A(2));
	hptr2 = hptr1;

	// 构建一个链表来测试
	Hptr<Node> head(new Node(1));
	head->next_ = Hptr<Node>(new Node(2));
	head->next_->next_ = Hptr<Node>(new Node(3));
}

int main()
{
	int* d_a;
	// 必须添加， 不添加kernel不输出
	hipMalloc((void**)&d_a, sizeof(int));

	hipError_t err;
	kernel << <1, 1 >> > ();//(d_a, d_b, d_c);
	err = hipGetLastError();
	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to launch kernel (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	hipFree(d_a);
	return 0;
}

#endif GPU_TEST
